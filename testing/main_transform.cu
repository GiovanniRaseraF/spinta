#include <transform.cuh>
#include <for_each.cuh>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

template <typename T>
class mark
{
    public:
    __host__ __device__ T operator()(T x){return x*x;}
};

template <typename T>
struct pres{
    public:
    __device__ void operator()(T x){printf("%d\n", x);}
};

int main(){
    std::vector<int> hh{10, 11, 12};

    thrust::device_vector<int> test{hh.begin(), hh.end()};
    mark<int> s;

    thrust::device_vector<int> res = spinta::parallel::transform(test.begin(), test.end(), s);
    spinta::parallel::for_each(res.begin(), res.end(), pres<int>());

    hipDeviceSynchronize();
}