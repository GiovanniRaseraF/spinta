#include <transform.cuh>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

template <typename T>
class mark
{
    public:
    __host__ __device__ T operator()(T x){return x+1;}
};

int main(){
    std::vector<int> hh{10, 11, 12};

    thrust::device_vector<int> test{hh.begin(), hh.end()};
    mark<int> s;
    spinta::parallel::transform(test.begin(), test.end(), s);

    hipDeviceSynchronize();
}