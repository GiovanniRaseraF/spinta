#include "hip/hip_runtime.h"
#include <spinta.cuh>

#include <vector>
#include <iostream>

template<typename T>
__global__ void setvalue(T* src, T* dest){
    int i = threadIdx.x;

    dest[i] = src[i];
}

int main(){
    // Testing RAII
    spinta::parallel::vector<int> myvet(10);
    spinta::parallel::vector<int> dest(10);

    setvalue<int><<<1,1>>>(myvet.gpu_ptr(), dest.gpu_ptr());

    hipDeviceSynchronize();
}
