#include <for_each.cuh>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

template <typename T>
class mark
{
    public:
    __host__ __device__ T operator()(T x){printf("%d\n", x*2);}
};

int main(){
    std::vector<int> hh{10, 11, 12};
    thrust::device_vector<int> test{hh.begin(), hh.end()};
    mark<int> s;

    spinta::parallel::for_each(test.begin(), test.end(), s);

    hipDeviceSynchronize();
}